// module load nvhpc/21.9
//nvc++ gpu.cu && ./a.out -N 500 -ll 15000


#include <hip/hip_runtime.h>
#include <string>
#include <fstream>
#include <vector>
#include <utility> // std::pair
#include <stdexcept> // std::runtime_error
#include <sstream> // std::stringstream
#include <iostream>
#include <math.h> 
#include <cstring>
#include <omp.h>
#include <chrono>

#define THREADS_PER_BLOCK 512

#define CUDA_DEBUG

#ifdef CUDA_DEBUG

#define CUDA_CHECK_ERROR(err)           \
if (err != hipSuccess) {          \
printf("Cuda error: %s\n", hipGetErrorString(err));    \
printf("Error in file: %s, line: %i\n", __FILE__, __LINE__);  \
}                 \

#else

#define CUDA_CHECK_ERROR(err)

#endif


std::vector<std::pair<std::string, std::vector<double>>> read_csv(std::string filename){
    std::vector<std::pair<std::string, std::vector<double>>> result;
    std::ifstream myFile(filename);
    if(!myFile.is_open()) throw std::runtime_error("Could not open file");
    std::string line, colname;
    double val;
    if(myFile.good())
    {
        // Extract the first line in the file
        std::getline(myFile, line);
        // Create a stringstream from line
        std::stringstream ss(line);
        // Extract each column name
        while(std::getline(ss, colname, ',')){
            // Initialize and add <colname, int vector> pairs to result
            result.push_back({colname, std::vector<double> {}});
        }
    }
    // Read data, line by line
    while(std::getline(myFile, line))
    {
        std::stringstream ss(line);
        int colIdx = 0;
        while(ss >> val){
            result.at(colIdx).second.push_back(val);
            if(ss.peek() == ',') ss.ignore();
            colIdx++;
        }
    }
    myFile.close();
    return result;
}

void print_map(double *rho, int N) {
        
        std::ofstream fout;
        fout.open("out.csv", std::fstream::out); //std::ios::app 
        
        if (!fout.is_open()) std::cout << "ошибка открытия out";
        else{
            for (int i = 0; i < N; ++i) fout<< "name,";
            fout<<std::endl;
            for (int i = 0; i < N; ++i)
            {
                for (int j = 0; j < N; ++j)
                {
                    std::cout.precision(17);
                    fout << std::fixed << rho[i*N + j] << ',';
                }
                fout<<std::endl;
            }
        }
        fout.close();
    }


__global__  void gpu(double *Dx_grid, double *Dz_grid, double *Drho, double *Dx_data, double *Dy_data, double *Dz_data, 
				double *Ddensity_data, int N, int len_data, double alpha, double ll){
    
    int ident = blockIdx.x * blockDim.x + threadIdx.x;
    int i = ident/N;
    int j = ident%N;
    Drho[ident] = 0.;
    for (int k = 0; k < len_data; ++k)
    {
        Drho[ident] += Ddensity_data[k] * exp( -alpha*sqrt(pow(Dx_grid[i] - Dx_data[k], 2.) +
                                                          pow(0          - Dy_data[k], 2.) + 
                                                          pow(Dz_grid[j] - Dz_data[k], 2.) )  );
    }
        
}



int main(int argc, char const *argv[]) {
    int N = 50;
    double ll = 1.;

    for (int i=1;i<argc;i++) {
        if (!strcmp(argv[i],"-N")) {
            N=atoi(argv[++i]);
            
        }
        else if (!strcmp(argv[i],"-ll")) {
            ll=(double)atof(argv[++i]);
        }
    }

    std::cout<< "N   " << N << std::endl;
    std::cout<< "ll   " << ll << std::endl;


    std::vector<std::pair<std::string, std::vector<double>>> data = read_csv("xyz270.csv");

    int len_data = data[0].second.size();
    std::cout<< "Num of str " << len_data << std::endl;

    
    double *x_data = new double [len_data];
    for (int i = 0; i < len_data; ++i) x_data[i] = data[0].second[i];
    double *y_data = new double [len_data];
    for (int i = 0; i < len_data; ++i) y_data[i] = data[1].second[i];
    double *z_data = new double [len_data];
    for (int i = 0; i < len_data; ++i) z_data[i] = data[2].second[i];
    double *density_data = new double [len_data];
    for (int i = 0; i < len_data; ++i) density_data[i] = data[3].second[i];

    double R = 3526229.;
    double alpha = R/10000000000000000.*ll;

    double *x_grid = new double [N];
    for (int i = 0; i < N; ++i) x_grid[i] = -R + (double)i/(N - 1)*2*R;

    double *z_grid = new double [N];
    for (int i = 0; i < N; ++i) z_grid[i] = -R + (double)i/(N - 1)*2*R;

    double *rho =  new double [N*N];
    for (int i = 0; i < N*N; ++i) rho[i] = 0.;

	double *Dz_grid, *Dx_grid, *Drho, *Dx_data, *Dy_data, *Dz_data, *Ddensity_data;
	
    hipMalloc((void**)&Dx_grid, sizeof(double)*N);	
	hipMalloc((void**)&Dz_grid, sizeof(double)*N);
	hipMalloc((void**)&Drho, sizeof(double)*N*N);	
	hipMalloc((void**)&Dx_data, sizeof(double)*len_data);
	hipMalloc((void**)&Dy_data, sizeof(double)*len_data);
	hipMalloc((void**)&Dz_data, sizeof(double)*len_data);	
	hipMalloc((void**)&Ddensity_data, sizeof(double)*len_data);


    const int nStream = 3; 
    hipStream_t stream[nStream];
    for (int i = 0; i < nStream; ++i ) hipStreamCreate ( &stream[i] );


    for (int i = 0; i < nStream; ++i ) // Копирование массивов с host на device
    {
        CUDA_CHECK_ERROR(hipMemcpyAsync(Dx_grid + i*N/nStream, x_grid + i*N/nStream, sizeof(double)*N/nStream, hipMemcpyHostToDevice, stream[i]));
        CUDA_CHECK_ERROR(hipMemcpyAsync(Dz_grid+ i*N/nStream, z_grid + i*N/nStream, sizeof(double)*N/nStream, hipMemcpyHostToDevice, stream[i]));
        CUDA_CHECK_ERROR(hipMemcpyAsync(Drho + i*N*N/nStream, rho+ i*N*N/nStream, sizeof(double)*N*N/nStream, hipMemcpyHostToDevice, stream[i]));
    	CUDA_CHECK_ERROR(hipMemcpyAsync(Dx_data + i*len_data/nStream, x_data + i*len_data/nStream, sizeof(double)*len_data/nStream, hipMemcpyHostToDevice, stream[i]));
    	CUDA_CHECK_ERROR(hipMemcpyAsync(Dy_data  + i*len_data/nStream, y_data + i*len_data/nStream, sizeof(double)*len_data/nStream, hipMemcpyHostToDevice, stream[i]));
    	CUDA_CHECK_ERROR(hipMemcpyAsync(Dz_data + i*len_data/nStream, z_data + i*len_data/nStream, sizeof(double)*len_data/nStream, hipMemcpyHostToDevice, stream[i]));
    	CUDA_CHECK_ERROR(hipMemcpyAsync(Ddensity_data + i*len_data/nStream, density_data + i*len_data/nStream, sizeof(double)*len_data/nStream, hipMemcpyHostToDevice, stream[i]));
    }

	float timerValueGPU;
	hipEvent_t start, stop;
    hipEventCreate ( &start);
    hipEventCreate ( &stop);
	hipEventRecord(start, 0);



    // for (int i = 0; i < nStream; ++i )
    // gpu <<< N*N/THREADS_PER_BLOCK, THREADS_PER_BLOCK, 0, stream[i] >>>(Dx_grid + i*N/nStream, Dz_grid+ i*N/nStream, Drho+ i*N*N/nStream, Dx_data, Dy_data, Dz_data, Ddensity_data, N, len_data, alpha, ll);
    gpu <<< N*N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (Dx_grid, Dz_grid, Drho, Dx_data, Dy_data, Dz_data, Ddensity_data, N, len_data, alpha, ll);
        

    hipEventRecord(stop,0);
   	hipEventSynchronize(stop);
    hipEventElapsedTime(&timerValueGPU, start, stop);

    for (int  i = 0; i < nStream; ++i ) // Копирование результат с device на host
        CUDA_CHECK_ERROR(hipMemcpyAsync(rho + i*N*N/nStream, Drho + i*N*N/nStream, sizeof(double)*N*N/nStream, hipMemcpyDeviceToHost, stream[i]));
    hipDeviceSynchronize ();

    
    printf ("\n GPU w/o shared memory 3 workers calculation time %f msec\n", timerValueGPU);
    std::cout << "alpha  " <<ll << " N   " << N << "   time    " << timerValueGPU << std::endl;

    std::ofstream fout;
    fout.open("logs.txt", std::fstream::app); //std::ios::app 
    if (!fout.is_open()) std::cout << "ошибка открытия logs";
    else fout<< "gpu "  << "  alpha  " <<ll << " N   " << N << "   time    "<< timerValueGPU << std::endl;
    fout.close();

    print_map(rho, N);
    for ( int i = 0; i < nStream; ++i ) hipStreamDestroy ( stream[i] );
    delete[] z_grid, x_grid, rho, x_data, y_data, z_data, density_data;
    hipFree( Dz_grid);
    hipFree(Dx_grid);
    hipFree(Drho);
    hipFree(Dx_data);
    hipFree(Dy_data);
    hipFree(Dz_data);
    hipFree(Ddensity_data);
    return 0;
}


